#include "hip/hip_runtime.h"
﻿#include "WarpDivergence.h"

__global__ void VectorInitializeWithWD(float* g_C)
{
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	float a, b;
	a = b = 0.0f;
	if (idx % 2 == 0)
	{
		a = 100.0f;
	}
	else
	{
		b = 200.0f;
	}
	g_C[idx] = a + b;
}

__global__ void VectorInitializeAcrossWD(float* g_C)
{
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	float a, b;
	a = b = 0.0f;
	if ((idx / warpSize) % 2 == 0)
	{
		a = 100.0f;
	}
	else
	{
		b = 200.0f;
	}
	g_C[idx] = a + b;
}

__global__ void VectorInitializeCompiler(float* g_C)
{
	const int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	float a, b;
	a = b = 0.0f;
	bool flag = (idx % 2 ==0);
	if (flag)
	{
		a = 100.0f;
	}
	if (!flag)
	{
		b = 200.0f;
	}
	g_C[idx] = a + b;
}

__host__ void VectorOperations(float* h_C)
{
	float* d_C;
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;

	float fElapsedTime;

	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);

	//Allocate device memory on the global memory
	hipMalloc((void**)&d_C, VectorSizeInBytes);


	//Kernel with Warp Divergence
	hipEventRecord(kernel_start);
	//Launch the Kernel with Warp Divergence
	VectorInitializeWithWD << <1, 32 >> > (d_C);
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "Kernel with Warp Divergence launch failed with error code: " << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop);
	hipMemcpy(h_C, d_C, VectorSizeInBytes, hipMemcpyDeviceToHost);

	cout << "Kernel with Warp Divergence = " << fElapsedTime << " msecs" << endl;

	//Kernel with Across Warp Divergence
	hipEventRecord(kernel_start);
	//Launch the Kernel with Across Warp Divergence
	VectorInitializeAcrossWD << <1, 32 >> > (d_C);
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "Kernel a Across Warp Divergence launch failed with error code: " << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop);
	hipMemcpy(h_C, d_C, VectorSizeInBytes, hipMemcpyDeviceToHost);

	cout << "Kernel with Across Warp Divergence = " << fElapsedTime << " msecs" << endl;

	//Kernel with Compiler implementation
	hipEventRecord(kernel_start);
	//Launch the Kernel with Compiler Implementation
	VectorInitializeCompiler << <1, 32 >> > (d_C);
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "Kernel with Compiler Implementation launch failed with error code: " << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop);
	hipMemcpy(h_C, d_C, VectorSizeInBytes, hipMemcpyDeviceToHost);

	cout << "Kernel with Compiler Implementation = " << fElapsedTime << " msecs" << endl;




	hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_stop);
	hipFree(d_C);
	hipDeviceReset();
}