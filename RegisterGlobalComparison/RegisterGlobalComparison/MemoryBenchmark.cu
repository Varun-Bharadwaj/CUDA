#include "hip/hip_runtime.h"
﻿#include "MemComparison.h"

__global__ void GlobalBM(int* g_Vector, int* g_Result, int size, int count)
{
	for (int i = 0; i < count; i++)
	{
		g_Result[i] = 0;
		for (int j = 0; j < size; j++)
		{
			g_Result[i] += g_Vector[j];
		}
	}
}

__global__ void RegisterBM(int* g_Vector, int* g_Result, int size, int count)
{0;
int temp;
	for (int i = 0; i < count; i++)
	{
		temp = 0;
		g_Result[i] = 0;
		for (int j = 0; j < size; j++)
		{
			temp += g_Vector[j];
		}
		g_Result[i] += temp;
	}
}

__host__ void MemoryBenchmark(int* pVector, int* pResult)
{
	int* d_Vector, * d_Result;
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	float fElapsedTime;

	//Create Event Objects
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);

	hipMalloc((void**)&d_Vector, DATA_SIZE_BYTES);
	hipMalloc((void**)&d_Result, DATA_SIZE_BYTES);

	hipMemcpy(d_Vector, pVector, DATA_SIZE_BYTES, hipMemcpyHostToDevice);

	//Global Memory
	hipEventRecord(kernel_start);
	//Launch the GlobalMemory Kernel
	GlobalBM << <1, 1 >> > (d_Vector, d_Result, SIZE, ITERATION_COUNT);
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "Global Memory Kernel launch failed with error code: " << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop);
	cout << "Global Memory Use: Elapsed Time (GPU) = " << fElapsedTime << " msecs" << endl;
	hipMemcpy(pResult, d_Result, DATA_SIZE_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_Vector);
	hipFree(d_Result);
	

	hipMalloc((void**)&d_Vector, DATA_SIZE_BYTES);
	hipMalloc((void**)&d_Result, DATA_SIZE_BYTES);

	hipMemcpy(d_Vector, pVector, DATA_SIZE_BYTES, hipMemcpyHostToDevice);
	

	//Register Memory
	hipEventRecord(kernel_start);
	//Launch the Register Memory Kernel
	RegisterBM << <1, 1 >> > (d_Vector, d_Result, SIZE, ITERATION_COUNT);
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cout << "Register Memory Kernel launch failed with error code: " << hipGetErrorString(cudaStatus) << endl;
		return;
	}
	hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop);
	cout << "Register Memory Use: Elapsed Time (GPU) = " << fElapsedTime << " msecs" << endl;
	hipMemcpy(pResult, d_Result, DATA_SIZE_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_Vector);
	hipFree(d_Result);
	hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_stop);
	hipDeviceReset();
}