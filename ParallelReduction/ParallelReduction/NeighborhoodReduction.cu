#include "hip/hip_runtime.h"
﻿#include "ParallelReduction.h"
#include "GPUErrors.h"

__global__ void NeighborhoodWithDivergence(float* g_Vector, float* g_PartialSum)
{
	//Save threadIdx.x on the register
	int tid = threadIdx.x;

	//Compute the global thread index
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);

	//Compute the local pointer to each block
	float* blockAddress = g_Vector + (blockIdx.x * blockDim.x);

	if (idx >= VECTOR_SIZE)
	{
		return;
	}

	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			blockAddress[tid] += blockAddress[tid + stride];
		}
		//synchronize within thread blocks
		__syncthreads();
	}
	if (tid == 0)
	{
		g_PartialSum[blockIdx.x] = blockAddress[0];
	}
}

__global__ void NeighborhoodWithLessDivergence(float* g_Vector, float* g_PartialSum)
{
	//Save threadIdx.x on the register
	int tid = threadIdx.x;

	//Compute the global thread index
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);

	float* blockAddress = g_Vector + (blockIdx.x * blockDim.x);

	if (idx >= VECTOR_SIZE)
	{
		return;
	}
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		int index = 2 * stride * tid;
		if (index < blockDim.x)
		{
			blockAddress[index] += blockAddress[index + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		g_PartialSum[blockIdx.x] = blockAddress[0];
	}
}

__host__ void OnNeighborhood(float* vectorTemp)
{
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	chrono::time_point<std::chrono::system_clock> start, end;

	float fElapsedTime;
	float fPartialReduceTime;

	HandleCUDAError(hipEventCreate(&kernel_start));
	HandleCUDAError(hipEventCreate(&kernel_stop));

	float* d_Vector;
	float* d_PartialSum;

	float* h_PartialSum;

	//Block and Thread Parameters
	dim3 block(256);
	dim3 grid((VECTOR_SIZE + block.x - 1) / block.x, 1);
	cout << "Neighborhood Implementations" << endl;
	cout << "\tThreads/Block: " << block.x << endl;
	cout << "\tBlocks/Grid: " << grid.x << endl;

	//The partial sums of each block
	h_PartialSum = new float[grid.x];

	//Allocate memory on the GPU to store the vector and partial sums
	HandleCUDAError(hipMalloc((void**)&d_Vector, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_PartialSum, (grid.x * sizeof(float))));

	//Copy the vector to the GPU from the host
	HandleCUDAError(hipMemcpy(d_Vector, vectorTemp, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));

	//Launch the Neighboorhood pairing kernel with Divergence
	HandleCUDAError(hipEventRecord(kernel_start));
	NeighborhoodWithDivergence << <grid, block >> > (d_Vector, d_PartialSum);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fPartialReduceTime, kernel_start, kernel_stop));
	
	//Copy the vector to the GPU from the host containing the sum of each block
	HandleCUDAError(hipMemcpy(h_PartialSum, d_PartialSum, (grid.x * sizeof(float)), hipMemcpyDeviceToHost));

	float sum = 0.0f;
	start = std::chrono::system_clock::now();
	for (int j = 0; j < grid.x; j++)
	{
		sum += h_PartialSum[j];
	}
	end = std::chrono::system_clock::now();
	std::chrono::duration<double> elasped_seconds = end - start;
	fElapsedTime = fPartialReduceTime + (elasped_seconds.count()*1000.0f);
	cout << "GPU Neighborhood Reduction with Divergence Execution time: " << fElapsedTime << " msecs" << endl;
	cout << "\t\tGPU Neighborhood Reduction: " << sum << endl;

	//Release Global Memory of d_Vector, and d_PartialSum
	HandleCUDAError(hipFree(d_Vector));
	HandleCUDAError(hipFree(d_PartialSum));
	//Reallocate Global Memory of d_Vector, and d_PartialSum
	HandleCUDAError(hipMalloc((void**)&d_Vector, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_PartialSum, (grid.x * sizeof(float))));
	//Copy the vector to the GPU from the host
	HandleCUDAError(hipMemcpy(d_Vector, vectorTemp, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));
	
	//Launch the Neighborhood pairing kernel with less divergence
	HandleCUDAError(hipEventRecord(kernel_start));
	NeighborhoodWithLessDivergence << <grid, block >> > (d_Vector, d_PartialSum);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fPartialReduceTime, kernel_start, kernel_stop));

	//Copy the vector to the GPU from the host containing the sum of each block
	HandleCUDAError(hipMemcpy(h_PartialSum, d_PartialSum, (grid.x * sizeof(float)), hipMemcpyDeviceToHost));
	sum = 0.0f;
	start = std::chrono::system_clock::now();
	for (int j = 0; j < grid.x; j++)
	{
		sum += h_PartialSum[j];
	}
	end = std::chrono::system_clock::now();
	elasped_seconds = end - start;
	fElapsedTime = fPartialReduceTime + (elasped_seconds.count() * 1000.0f);
	cout << "GPU Neighborhood Reduction with Less Divergence Execution time: " << fElapsedTime << " msecs" << endl;
	cout << "\t\tGPU Neighborhood Reduction: " << sum << endl;

	HandleCUDAError(hipFree(d_Vector));
	HandleCUDAError(hipFree(d_PartialSum));
	HandleCUDAError(hipEventDestroy(kernel_start));
	HandleCUDAError(hipEventDestroy(kernel_stop));
	HandleCUDAError(hipDeviceReset());
}