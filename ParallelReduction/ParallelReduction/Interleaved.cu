#include "hip/hip_runtime.h"
﻿#include "ParallelReduction.h"
#include "GPUErrors.h"

__global__ void OnInterleavedWithoutUnroll(float* g_Vector, float* g_PartialSum)
{
	//Save threadIdx.x on the register
	int tid = threadIdx.x;

	//Compute the global thread index
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	float* blockAddress = g_Vector + (blockIdx.x * blockDim.x);

	if (idx >= VECTOR_SIZE)
	{
		return;
	}

	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		g_PartialSum[blockIdx.x] = blockAddress[0];
	}
}

__host__ void OnInterleaved(float* vectorTemp)
{
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	chrono::time_point<std::chrono::system_clock> start, end;

	float fElapsedTime;
	float fPartialReduceTime;

	HandleCUDAError(hipEventCreate(&kernel_start));
	HandleCUDAError(hipEventCreate(&kernel_stop));

	float* d_Vector;
	float* d_PartialSum;

	float* h_PartialSum;

	//Block and Thread Parameters
	dim3 block(256);
	dim3 grid((VECTOR_SIZE + block.x - 1) / block.x, 1);
	cout << "Neighborhood Implementations" << endl;
	cout << "\tThreads/Block: " << block.x << endl;
	cout << "\tBlocks/Grid: " << grid.x << endl;

	//The partial sums of each block
	h_PartialSum = new float[grid.x];

	//Allocate memory on the GPU to store the vector and partial sums
	HandleCUDAError(hipMalloc((void**)&d_Vector, VECTOR_SIZE_IN_BYTES));
	HandleCUDAError(hipMalloc((void**)&d_PartialSum, (grid.x * sizeof(float))));

	//Copy the vector to the GPU from the host
	HandleCUDAError(hipMemcpy(d_Vector, vectorTemp, VECTOR_SIZE_IN_BYTES, hipMemcpyHostToDevice));

	//Launch the Interleaved pairing kernel without unrolling
	HandleCUDAError(hipEventRecord(kernel_start));
	OnInterleavedWithoutUnroll << <grid, block >> > (d_Vector, d_PartialSum);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fPartialReduceTime, kernel_start, kernel_stop));

	//Copy the vector to the GPU from the host containing the sum of each block
	HandleCUDAError(hipMemcpy(h_PartialSum, d_PartialSum, (grid.x * sizeof(float)), hipMemcpyDeviceToHost));

	float sum = 0.0f;
	start = std::chrono::system_clock::now();
	for (int j = 0; j < grid.x; j++)
	{
		sum += h_PartialSum[j];
	}
	end = std::chrono::system_clock::now();
	std::chrono::duration<double> elasped_seconds = end - start;
	fElapsedTime = fPartialReduceTime + (elasped_seconds.count() * 1000.0f);
	cout << "GPU Interleaved Reduction without unroll Execution time: " << fElapsedTime << " msecs" << endl;
	cout << "\t\tGPU Interleaved Reduction: " << sum << endl;

	HandleCUDAError(hipFree(d_Vector));
	HandleCUDAError(hipFree(d_PartialSum));
	HandleCUDAError(hipEventDestroy(kernel_start));
	HandleCUDAError(hipEventDestroy(kernel_stop));
	HandleCUDAError(hipDeviceReset());
}