#include "hip/hip_runtime.h"
#include "MatrixTranspose.h"
#include "GPUErrors.h"

__global__ void CopyRowWise(float* g_Matrix, float* g_MatrixCopy, int ny, int nx)
{
	unsigned int ix = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int iy = threadIdx.y + (blockIdx.y * blockDim.y);

	if (ix < nx && iy < ny)
	{
		g_MatrixCopy[iy * nx + ix] = g_Matrix[iy * nx + ix];
	}
}

__global__ void CopyColWise(float* g_Matrix, float* g_MatrixCopy, int ny, int nx)
{
	unsigned int ix = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned int iy = threadIdx.y + (blockIdx.y * blockDim.y);

	if (ix < nx && iy < ny)
	{
		g_MatrixCopy[ix * ny + iy] = g_Matrix[ix * ny + iy];
	}

}

__host__ void PerformanceBounds(float *h_Matrix, int ny, int nx)
{
	float *d_Matrix;
	float *d_MatrixCopy;

	float *h_MatrixCopy = new float[ny*nx];
	const int MatrixSizeInBytes = ny * nx * sizeof(float);

	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	float fElapsedTime;

	HandleCUDAError(hipEventCreate(&kernel_start));
	HandleCUDAError(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	HandleCUDAError(hipMalloc((void**)&d_Matrix, MatrixSizeInBytes));
	HandleCUDAError(hipMalloc((void**)&d_MatrixCopy, MatrixSizeInBytes));

	//transfer data from CPU Memory to GPU Memory
	HandleCUDAError(hipMemcpy(d_Matrix, h_Matrix, MatrixSizeInBytes, hipMemcpyHostToDevice));

	//Block and Grid Parameters
	int dimx = 16;
	int dimy = 16;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	cout << "2D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
	cout << "\tNumber of Blocks along Y dimension: " << grid.y << endl;
	cout << "2D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block.x << endl;
	cout << "\tNumber of threads along Y dimension: " << block.y << endl;

	//Matrix Copy Row wise - Coalesced Access
	HandleCUDAError(hipEventRecord(kernel_start));
	CopyRowWise << <grid, block >> > (d_Matrix, d_MatrixCopy, ny, nx);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop));
	HandleCUDAError(hipMemcpy(h_MatrixCopy, d_MatrixCopy, MatrixSizeInBytes, hipMemcpyDeviceToHost));

	cout << "Row Wise: Matrix Copy Elapsed Time = " << fElapsedTime << " msecs" << endl;

	//Matrix Copy Column wise - Stride Access
	HandleCUDAError(hipEventRecord(kernel_start));
	CopyColWise << <grid, block >> > (d_Matrix, d_MatrixCopy, ny, nx);
	HandleCUDAError(hipEventRecord(kernel_stop));
	HandleCUDAError(hipEventSynchronize(kernel_stop));
	GetCUDARunTimeError();
	HandleCUDAError(hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop));
	HandleCUDAError(hipMemcpy(h_MatrixCopy, d_MatrixCopy, MatrixSizeInBytes, hipMemcpyDeviceToHost));

	cout << "Column Wise: Matrix Copy Elapsed Time = " << fElapsedTime << " msecs" << endl;

	delete[] h_MatrixCopy;
	HandleCUDAError(hipFree(d_Matrix));
	HandleCUDAError(hipFree(d_MatrixCopy));
	HandleCUDAError(hipEventDestroy(kernel_start));
	HandleCUDAError(hipEventDestroy(kernel_stop));
	HandleCUDAError(hipDeviceReset());
}

